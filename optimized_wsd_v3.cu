#include "hip/hip_runtime.h"
/* 
* Simplified Sense Word Disambiguation algorithm written by Ahmed Siddiqui and
* Jordan Kirchner based off of Michael Lesk's simplified algorithm
* (https://en.wikipedia.org/wiki/Lesk_algorithm#Simplified_Lesk_algorithm)
*/

/*

Algorithm: 
function SIMPLIFIED LESK(word,sentence) returns best sense of word
    best-sense <- most frequent sense for word
    max-overlap <- 0
    context <- set of words in sentence
    for each sense in senses of word do
        signature <- set of words in the gloss and examples of sense
        overlap <- COMPUTEOVERLAP (signature,context)
        if overlap > max-overlap then
            max-overlap <- overlap
            best-sense <- sense
end return (best-sense)

*/

#include <chrono>
#include <string>
#include <set>
#include <vector>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <stdio.h>
#include "picojson.h"
#include "wsd_v3.cuh"

using namespace picojson;
using namespace std;

#define MAX_SENSE_SIZE 300


string remove_punctuation(string str) {
    string result;
    std::remove_copy_if(str.begin(), str.end(),            
                std::back_inserter(result), //Store output           
                ::ispunct);

    return result;
}

int hash_string(string str) {
    /* 
    Need to lowercase first, and then hash it. 
    */
    std::transform(str.begin(), str.end(), str.begin(), ::tolower);
    std::hash<std::string> hasher;
    return hasher(str);
}


__global__ 
void compute_overlap(const int *senses, const int *context, int *overlaps, int senses_size, int context_size) {
    int sense_index = (threadIdx.x + blockIdx.x * blockDim.x) * MAX_SENSE_SIZE;
    if (sense_index < senses_size) {
        for (int i = sense_index; i < sense_index + MAX_SENSE_SIZE; i++) {
            if (senses[i] == -1) {
                break;
            }
            for (int j = 0; j < context_size; j++) {
                overlaps[sense_index/MAX_SENSE_SIZE] += (senses[i] == context[j]);
            }
        }
    }
    __syncthreads();
}

vector<string> get_all_senses(string word) {
    /* 
    This function will query dictionary.json and get the definition of the
    word. It will then parse through the definition and get all the senses.
    It will then store all those senes in the given vector: all_senses
    */
    // read a JSON file
    string dictionary_name = "final_dictionary/";
    dictionary_name += word[0];
    if (word[1] != '\0')
        dictionary_name += word[1];
    dictionary_name += ".json";

    std::ifstream i(dictionary_name);
    value v;
    i >> v;

    std::vector<std::string> items; 
    std::vector<picojson::value> tmp = v.get(word).get<picojson::array>();
    for (int i = 0; i < tmp.size(); i++) {
        items.push_back(tmp[i].get<std::string>());
    }

    return items;
}

vector<int> get_word_set(string word, string sentence) {
    vector<int> words = tokenize_string(sentence);
    words.erase(std::remove(words.begin(), words.end(), hash_string(word)), words.end());
    return words;
}

vector<int> tokenize_string(string sentence) {
    stringstream stream(sentence);
    set<int> words;
    string tmp;
    while (getline(stream, tmp, ' ')) {
        words.insert(hash_string(remove_punctuation(tmp)));
    }
    
    return vector<int>(words.begin(), words.end());
}

// We need to turn compute overlap into a kernel function.
// We need to get rid of the for loop, create dev_senses, dev_context, 
// and send in a results array to compute_overlap, 
// hipMalloc, hipMemcpy, hipFree them up, then 
// find the max result from that array 
string simplified_wsd(string word, string sentence) {
    string best_sense;
    int max_overlap = -1;
    vector<int> context = get_word_set(word, sentence);// This is the set of words in a sentence excluding the word itself.
    vector<string> all_senses = get_all_senses(word);
    vector<int> hashed_sense_tokens; 
    for (int i = 0; i < all_senses.size(); i++) {
        // for every sense, we want to tokenize the string, then for every token, we want to add that to a new vector. 
        vector<int> tokens = tokenize_string(all_senses[i]);
        for (int j = 0; j < tokens.size(); j++) {
            hashed_sense_tokens.push_back(tokens[j]);
        }
        while (hashed_sense_tokens.size() % MAX_SENSE_SIZE != 0) {
            hashed_sense_tokens.push_back(-1);
        }
    }
    int *dev_senses;
    int *dev_context;
    int *dev_results;
    int overlaps[all_senses.size()];
    int const block_size = 960;
    hipMalloc((void **) &dev_senses, all_senses.size() * MAX_SENSE_SIZE * sizeof(hashed_sense_tokens[0]));
    hipMalloc((void **) &dev_context, context.size() * sizeof(context[0]));
    hipMalloc((void **) &dev_results, all_senses.size() * sizeof(int));

    hipMemcpy(dev_senses, &hashed_sense_tokens[0], hashed_sense_tokens.size() * sizeof(hashed_sense_tokens[0]), hipMemcpyHostToDevice);
    hipMemcpy(dev_context, &context[0], context.size() * sizeof(context[0]), hipMemcpyHostToDevice);
    auto const num_blocks = ceil(all_senses.size() / static_cast<float>(block_size));
    auto const start = chrono::steady_clock::now();

    compute_overlap<<<num_blocks, block_size>>>(dev_senses, dev_context, dev_results, all_senses.size() * MAX_SENSE_SIZE, context.size());
    hipMemcpy(overlaps, dev_results, all_senses.size() * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < all_senses.size(); i++) {
        if (overlaps[i] > max_overlap) {
            best_sense = all_senses[i];
            max_overlap = overlaps[i];
        }
    }

    auto const end = chrono::steady_clock::now();

    hipFree(dev_results);
    hipFree(dev_context);
    hipFree(dev_senses);
    
    cout << "Time to run compute overlap was: " << chrono::duration <double, milli> (end - start).count() << " ms" << endl;

    return best_sense;
}

int main(int argc, char ** argv)
{
    /*
     cout << "Find the best sense of the word 'stock' in the following sentence:\n\tI'm expecting to make a lot of money from the stocks I'm investing in using my bank account.\n";
     cout << "The best sense of the word stock in our example is:\n" << simplified_wsd("stock", "I'm expecting to make a lot of money from the stocks I'm investing in using my bank account.") << "\n";
     */
    
    // auto start = chrono::steady_clock::now();
    
    simplified_wsd("set", "It was a great day of tennis. Game, set, match. Lorem ipsum dolor sit amet, consectetur adipiscing elit. Praesent in nunc ac metus condimentum volutpat. Vivamus ornare orci nec congue cursus. Curabitur ultricies sem eget metus lacinia, faucibus sagittis odio porta. Cras mollis malesuada justo sit amet fringilla. Fusce dolor est, pulvinar sit amet eleifend sollicitudin, pulvinar vel libero. Nunc nulla diam, vulputate in cursus in, faucibus non arcu. Phasellus semper tempus maximus. Mauris turpis velit, pulvinar eget tempor sit amet, pharetra id ante. Pellentesque blandit ac magna vel condimentum. Aenean aliquet pellentesque turpis sit amet eleifend. In dolor ligula, tempor non lacus euismod, blandit iaculis ex.Vestibulum maximus facilisis condimentum. Maecenas pretium malesuada nisi. Duis nec turpis vel justo mattis facilisis. Sed vulputate nulla in diam ultricies mollis vitae non dui. Nam nec tellus interdum lorem suscipit mollis. Aenean et nisl euismod, blandit augue sit amet, pellentesque tellus. Curabitur maximus mollis nibh eget mattis. Duis varius hendrerit euismod. Ut ac ipsum imperdiet, pretium purus et, aliquet risus. Nam nisi mauris, maximus ac venenatis a, lacinia ut nibh. Praesent id ligula volutpat, mollis orci ac, iaculis ligula. Aliquam pretium tortor posuere urna vestibulum efficitur. Suspendisse aliquet diam nec lectus consectetur, ut condimentum est consectetur. Vivamus blandit interdum leo quis tincidunt.Suspendisse potenti. Duis volutpat justo at eros lacinia feugiat. In sit amet volutpat turpis. Nulla tempor imperdiet elit, in vulputate urna placerat eu. Integer congue dolor nisi, vitae sagittis purus eleifend id. Pellentesque habitant morbi tristique senectus et netus et malesuada fames ac turpis egestas. Ut libero arcu, tempus sollicitudin leo a, euismod maximus ex. Nunc varius aliquet metus, a fermentum ligula sagittis vitae.Integer hendrerit nunc eget urna rutrum, ut viverra ligula tincidunt. Nulla viverra orci lectus, ut varius erat viverra ut. Proin ac lacus congue, bibendum lorem ut, placerat ex. Mauris a rhoncus sapien. Pellentesque vel ultricies tortor, at hendrerit metus. Nunc cursus commodo facilisis. In tempor neque ut condimentum cursus. Mauris ut erat facilisis, pulvinar lectus vitae, euismod erat. Nullam feugiat sollicitudin finibus. Cras ante quam, tincidunt sed ultrices quis, rutrum vel nibh. Suspendisse eu velit vitae tellus porttitor mollis a eget lectus. Sed vel nisl sed dui elementum feugiat. Fusce fermentum ex vel mi pellentesque, a ultrices erat vehicula. Aenean auctor erat eu justo tristique, eget fringilla augue commodo.Phasellus vel risus ante. Cras tristique neque velit, et suscipit metus maximus ut. Proin quis sem purus. Maecenas pharetra at felis eget pellentesque. Fusce blandit libero eget massa efficitur semper. Nulla eget euismod enim. Sed id enim ac eros luctus sodales in maximus mi.");
    
    // auto end = chrono::steady_clock::now();
    // auto diff = end - start;
    
    // cout << "Total time to run was: " << chrono::duration <double, milli> (diff).count() << " ms" << endl;
    
     return 0;
}
