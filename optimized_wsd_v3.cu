#include "hip/hip_runtime.h"
/* 
* Simplified Sense Word Disambiguation algorithm written by Ahmed Siddiqui and
* Jordan Kirchner based off of Michael Lesk's simplified algorithm
* (https://en.wikipedia.org/wiki/Lesk_algorithm#Simplified_Lesk_algorithm)
*/

/*

Algorithm: 
function SIMPLIFIED LESK(word,sentence) returns best sense of word
    best-sense <- most frequent sense for word
    max-overlap <- 0
    context <- set of words in sentence
    for each sense in senses of word do
        signature <- set of words in the gloss and examples of sense
        overlap <- COMPUTEOVERLAP (signature,context)
        if overlap > max-overlap then
            max-overlap <- overlap
            best-sense <- sense
end return (best-sense)

*/

#include <chrono>
#include <string>
#include <set>
#include <vector>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <stdio.h>
#include "picojson.h"
#include "wsd_v3.cuh"

using namespace picojson;
using namespace std;

#define MAX_SENSE_SIZE 100


string remove_punctuation(string str) {
    string result;
    std::remove_copy_if(str.begin(), str.end(),            
                std::back_inserter(result), //Store output           
                ::ispunct);

    return result;
}

int hash_string(string str) {
    /* 
    Need to lowercase first, and then hash it. 
    */
    std::transform(str.begin(), str.end(), str.begin(), ::tolower);
    std::hash<std::string> hasher;
    return hasher(str);
}


__global__ 
void compute_overlap(const int *senses, const int *context, int *overlaps, int senses_size, int context_size) {
    int sense_index = (threadIdx.x + blockIdx.x * blockDim.x) * MAX_SENSE_SIZE;
    if (sense_index < senses_size) {
        for (int i = sense_index; i < sense_index + MAX_SENSE_SIZE; i++) {
            if (senses[i] == -1) {
                break;
            }
            for (int j = 0; j < context_size; j++) {
                overlaps[sense_index/MAX_SENSE_SIZE] += (senses[i] == context[j]);
            }
        }
    }
}

vector<string> get_all_senses(string word) {
    /* 
    This function will query dictionary.json and get the definition of the
    word. It will then parse through the definition and get all the senses.
    It will then store all those senes in the given vector: all_senses
    */
    // read a JSON file
    string dictionary_name = "final_dictionary/";
    dictionary_name += word[0];
    if (word[1] != '\0')
        dictionary_name += word[1];
    dictionary_name += ".json";

    std::ifstream i(dictionary_name);
    value v;
    i >> v;

    std::vector<std::string> items; 
    std::vector<picojson::value> tmp = v.get(word).get<picojson::array>();
    for (int i = 0; i < tmp.size(); i++) {
        items.push_back(tmp[i].get<std::string>());
    }

    return items;
}

vector<int> get_word_set(string word, string sentence) {
    vector<int> words = tokenize_string(sentence);
    words.erase(std::remove(words.begin(), words.end(), hash_string(word)), words.end());
    return words;
}

vector<int> tokenize_string(string sentence) {
    stringstream stream(sentence);
    set<int> words;
    string tmp;
    while (getline(stream, tmp, ' ')) {
        words.insert(hash_string(remove_punctuation(tmp)));
    }
    
    return vector<int>(words.begin(), words.end());
}

// We need to turn compute overlap into a kernel function.
// We need to get rid of the for loop, create dev_senses, dev_context, 
// and send in a results array to compute_overlap, 
// hipMalloc, hipMemcpy, hipFree them up, then 
// find the max result from that array 
string simplified_wsd(string word, string sentence) {
    string best_sense;
    int max_overlap = 0;
    vector<int> context = get_word_set(word, sentence);// This is the set of words in a sentence excluding the word itself.
    vector<string> all_senses = get_all_senses(word);
    vector<int> hashed_sense_tokens; 
    for (int i = 0; i < all_senses.size(); i++) {
        // for every sense, we want to tokenize the string, then for every token, we want to add that to a new vector. 
        vector<int> tokens = tokenize_string(all_senses[i]);
        for (int j = 0; j < tokens.size(); j++) {
            hashed_sense_tokens.push_back(tokens[j]);
        }
        while (hashed_sense_tokens.size() % MAX_SENSE_SIZE != 0) {
            hashed_sense_tokens.push_back(-1);
        }
    }
    int *dev_senses;
    int *dev_context;
    int *dev_results;
    int overlaps[all_senses.size()];
    int const block_size = 128;
    hipMalloc((void **) &dev_senses, all_senses.size() * MAX_SENSE_SIZE * sizeof(hashed_sense_tokens[0]));
    hipMalloc((void **) &dev_context, context.size() * sizeof(context[0]));
    hipMalloc((void **) &dev_results, all_senses.size() * sizeof(int));

    hipMemcpy(dev_senses, &hashed_sense_tokens[0], hashed_sense_tokens.size() * sizeof(hashed_sense_tokens[0]), hipMemcpyHostToDevice);
    hipMemcpy(dev_context, &context[0], context.size() * sizeof(context[0]), hipMemcpyHostToDevice);
    auto const num_blocks = ceil(all_senses.size() / static_cast<float>(block_size));
    auto const start = chrono::steady_clock::now();

    compute_overlap<<<num_blocks, block_size>>>(dev_senses, dev_context, dev_results, all_senses.size() * MAX_SENSE_SIZE, context.size());
    hipMemcpy(overlaps, dev_results, all_senses.size() * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < all_senses.size(); i++) {
        if (overlaps[i] > max_overlap) {
            best_sense = all_senses[i];
            max_overlap = overlaps[i];
        }
    }

    auto const end = chrono::steady_clock::now();

    hipFree(dev_results);
    hipFree(dev_context);
    hipFree(dev_senses);
    
    cout << "Time to run compute overlap was: " << chrono::duration <double, milli> (end - start).count() << " ms" << endl;
    cout << "best sense is " << best_sense << endl;
    return best_sense;
}

int main(int argc, char ** argv)
{
    /*
     cout << "Find the best sense of the word 'stock' in the following sentence:\n\tI'm expecting to make a lot of money from the stocks I'm investing in using my bank account.\n";
     cout << "The best sense of the word stock in our example is:\n" << simplified_wsd("stock", "I'm expecting to make a lot of money from the stocks I'm investing in using my bank account.") << "\n";
     */
    
    // auto start = chrono::steady_clock::now();
    
    simplified_wsd("set", "My opponent won the first set in our tennis game.");
    
    // auto end = chrono::steady_clock::now();
    // auto diff = end - start;
    
    // cout << "Total time to run was: " << chrono::duration <double, milli> (diff).count() << " ms" << endl;
    
     return 0;
}
